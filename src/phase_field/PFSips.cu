# include <iostream>    // endl
# include <fstream>     // for ofstream
# include <string>      // for string
# include <sstream>     // for stringstream
# include <math.h>
# include "PFSips.h"
# include "PFSipsKernels.h"
# include "../utils/cudaErrorMacros.h" // for cudaCheckErrors & cudaCheckAsyncErrors


using std::string;
using std::stringstream;
using std::cout;
using std::endl;
using std::ofstream;

// -------------------------------------------------------------------------
// Constructor:
// -------------------------------------------------------------------------

PFSips::PFSips(const GetPot& input_params)
    : rng(1234)
{

    // ---------------------------------------
    // Assign variables from 'input_params':
    // ---------------------------------------

    nx = input_params("Domain/nx",1);
    ny = input_params("Domain/ny",1);
    nz = input_params("Domain/nz",1);
    nxyz = nx*ny*nz;
    dx = input_params("Domain/dx",1.0);
    dy = input_params("Domain/dy",1.0);
    dz = input_params("Domain/dz",1.0);
    dt = input_params("Time/dt",1.0);
    bx = input_params("PFSips/bx",0);
    by = input_params("PFSips/by",1);
    bz = input_params("PFSips/bz",1);
    numSteps = input_params("Time/nstep",1);
    co = input_params("PFSips/co",0.20);
    M = input_params("PFSips/M",1.0);
    mobReSize = input_params("PFSips/mobReSize",0.35);
    kap = input_params("PFSips/kap",1.0);
    water_CB = input_params("PFSips/water_CB",1.0);
    NS_in_dope = input_params("PFSips/NS_in_dope",0.0);
    mobReSize = input_params("PFSips/mobReSize",0.35);
    chiPS = input_params("PFSips/chiPS",0.034);
    chiPN = input_params("PFSips/chiPN",1.5);
    phiCutoff = input_params("PFSips/phiCutoff",0.75);
    N = input_params("PFSips/N",100.0);
    A = input_params("PFSips/A",1.0);
    Tinit = input_params("PFSips/Tinit",298.0);
    Tcast = input_params("PFSips/Tcast",298.0);
    noiseStr = input_params("PFSips/noiseStr",0.1);
    D0 = input_params("PFSips/D0",1.0);
    Dw = input_params("PFSips/Dw",1.0);
    nu = input_params("PFSips/nu",1.0);
    nuDw = input_params("PFSips/nuDw",1.0);
    gamma = input_params("PFSips/gamma",1.0);
    gammaDw = input_params("PFSips/gammaDw",1.0);
    Mweight = input_params("PFSips/Mweight",100.0);
    Mvolume = input_params("PFSips/Mvolume",0.1);
    numOutputs = input_params("Output/numOutputs",1);
    outInterval = numSteps/numOutputs;
    // ---------------------------------------
    // Set up cuda kernel launch variables:
    // ---------------------------------------

    blockSize.x = input_params("GPU/blockSize.x",0);
    blockSize.y = input_params("GPU/blockSize.y",0);
    blockSize.z = input_params("GPU/blockSize.z",0);

    // set default kernel launch parameters
    if(blockSize.x == 0) blockSize.x = 32;
    if(blockSize.y == 0) blockSize.y = 32;
    if(blockSize.z == 0) blockSize.z = 1;

    // calculate the number of blocks to be used (3-D block grid)
    int totalBlockSize = blockSize.x*blockSize.y*blockSize.z;
    blocks.x = (nx + blockSize.x - 1)/blockSize.x;
    blocks.y = (ny + blockSize.y - 1)/blockSize.y;
    blocks.z = (nz + blockSize.z - 1)/blockSize.z;

    // perform some assumption checking
    int numBlocks = blocks.x*blocks.y*blocks.z;
    int totalNumThreads = numBlocks*totalBlockSize;
    if(totalNumThreads < nxyz)
        throw "GPU Kernel Launch setup lacks sufficient threads!\n";
    if(totalBlockSize > 1024)
        throw "Total number of threads per block exceeds 1024";

}



// -------------------------------------------------------------------------
// Destructor:
// -------------------------------------------------------------------------

PFSips::~PFSips()
{

    // ----------------------------------------
    // free up device memory:
    // ----------------------------------------

    hipFree(c_d);
    hipFree(df_d);
    hipFree(Mob_d);
    hipFree(w_d);
    hipFree(muNS_d);
    hipFree(nonUniformLap_d);
    hipFree(cpyBuff_d);
    hipFree(devState);
}



// -------------------------------------------------------------------------
// Initialize system:
// -------------------------------------------------------------------------

void PFSips::initSystem()
{
		
    // ----------------------------------------
    // Initialize concentration fields:
    // ----------------------------------------
    srand(time(NULL));      // setting the seed  
    double r = 0.0;
    for(size_t i=0;i<nxyz;i++) {
        r = (double)rand()/RAND_MAX;
        // initialize polymer phase
        c.push_back(co + 0.1*(r-0.5));
        // initialize nonsolvent phase
        water.push_back(NS_in_dope);
    }

    // ----------------------------------------
    // Allocate memory on device and copy data
    // and copy data from host to device
    // ----------------------------------------

    // allocate memory on device
    size = nxyz*sizeof(double);
    // allocate polymer species
    hipMalloc((void**) &c_d,size);
    cudaCheckErrors("hipMalloc fail");
    // allocate space for laplacian
    hipMalloc((void**) &df_d,size);
    cudaCheckErrors("hipMalloc fail");
    // allocate water concentration
    hipMalloc((void**) &w_d,size);
    cudaCheckErrors("hipMalloc fail");
    // allocate space for laplacian
    hipMalloc((void**) &muNS_d,size);
    cudaCheckErrors("hipMalloc fail");
    // copy buffer
    hipMalloc((void**) &cpyBuff_d,size);
    cudaCheckErrors("hipMalloc fail");
    // allocate mobility
    hipMalloc((void**) &Mob_d,size);
    cudaCheckErrors("hipMalloc fail");
    // allocate nonuniform laplacian for mobility 
    // and water diffusion coefficient
    hipMalloc((void**) &nonUniformLap_d,size);
    cudaCheckErrors("hipMalloc fail");
    // allocate memory for cuRAND state
    hipMalloc((void**) &devState,nxyz*sizeof(hiprandState));
    cudaCheckErrors("hipMalloc fail");
    // copy concentration and water array to device
    hipMemcpy(c_d,&c[0],size,hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D fail");
    hipMemcpy(w_d,&water[0],size,hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D fail");
    
    // ----------------------------------------
    // Initialize thermal fluctuations of
    // polymer concentration
    // ----------------------------------------
    
    init_cuRAND<<<blocks,blockSize>>>(time(NULL),devState,nx,ny,nz);
    
}



// -------------------------------------------------------------------------
// Take one step forward in time:
// -------------------------------------------------------------------------

void PFSips::computeInterval(int interval)
{

    // ----------------------------------------
    //	Set the time step:
    // ----------------------------------------

    current_step = interval*outInterval;

    // ----------------------------------------
    //	Evolve system by solving CH equation:
    // ----------------------------------------

    for(size_t i=0;i<outInterval;i++)
    {
        // calculate the laplacian of c_d and store in df_d
        calculateLapBoundaries<<<blocks,blockSize>>>(c_d,df_d,nx,ny,nz,dx,bx,by,bz); 
        cudaCheckAsyncErrors("calculateLap polymer kernel fail");
        hipDeviceSynchronize();
        
        // calculate the chemical potential and store in df_d
        calculateChemPotFH<<<blocks,blockSize>>>(c_d,w_d,df_d,kap,A,chiPS,chiPN,N,nx,ny,nz,current_step,dt);
        cudaCheckAsyncErrors("calculateChemPotFH kernel fail");
        hipDeviceSynchronize();
        
        // calculate mobility and store it in Mob_d
        calculateMobility<<<blocks,blockSize>>>(c_d,Mob_d,M,mobReSize,nx,ny,nz,phiCutoff,N,gamma,nu,D0,Mweight,Mvolume,Tcast);
        cudaCheckAsyncErrors("calculateMobility kernel fail");
        hipDeviceSynchronize();

        // calculate the laplacian of the chemical potential, then update c_d
        // using an Euler update
        lapChemPotAndUpdateBoundaries<<<blocks,blockSize>>>(c_d,df_d,Mob_d,nonUniformLap_d, dt,nx,ny,nz,dx,bx,by,bz);
        cudaCheckAsyncErrors("lapChemPotAndUpdateBoundaries kernel fail");
        hipDeviceSynchronize();
        
        // calculate mu for Nonsolvent diffusion
        calculate_muNS<<<blocks,blockSize>>>(w_d,c_d,muNS_d,Mob_d,Dw,water_CB,gammaDw,nuDw,Mweight,Mvolume,nx,ny,nz);
        cudaCheckAsyncErrors('calculate muNS kernel fail');
        hipDeviceSynchronize();
        
        // calculate laplacian for diffusing water
        calculateLapBoundaries_muNS<<<blocks,blockSize>>>(df_d,muNS_d,nx,ny,nz,dx,bx,by,bz);
        cudaCheckAsyncErrors('calculateLap water kernel fail');    
        hipDeviceSynchronize();
        
        // calculate nonuniform laplacian for diffusion
        calculateNonUniformLapBoundaries_muNS<<<blocks,blockSize>>>(muNS_d,Mob_d,nonUniformLap_d,nx,ny,nz,dx,bx,by,bz);
        cudaCheckAsyncErrors('calculateNonUniformLap muNS kernel fail');
        hipDeviceSynchronize();
        
        // euler update water diffusing
        update_water<<<blocks,blockSize>>>(w_d,df_d,Mob_d,nonUniformLap_d,dt,nx,ny,nz,dx,bx,by,bz);
        cudaCheckAsyncErrors("updateWater kernel fail");
        hipDeviceSynchronize();
        
        // add thermal fluctuations of polymer concentration
        addNoise<<<blocks,blockSize>>>(c_d, nx, ny, nz, dt, current_step, water_CB, phiCutoff, devState);
        cudaCheckAsyncErrors("addNoise kernel fail");
        hipDeviceSynchronize(); 
    }

    // ----------------------------------------
    //	Copy data back to host for writing:
    // ----------------------------------------
    
    // polymer concentration
    populateCopyBufferSIPS<<<blocks,blockSize>>>(c_d,cpyBuff_d,nx,ny,nz);
    hipMemcpyAsync(&c[0],c_d,size,hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpyAsync D2H fail");
    hipDeviceSynchronize();
    // nonsolvent concentration
    populateCopyBufferSIPS<<<blocks,blockSize>>>(w_d,cpyBuff_d,nx,ny,nz);
    hipMemcpyAsync(&water[0],w_d,size,hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpyAsync D2H fail");
    hipDeviceSynchronize();
}



// -------------------------------------------------------------------------
// Write output:
// -------------------------------------------------------------------------

void PFSips::writeOutput(int step)
{

    // -----------------------------------
    // Define the file location and name:
    // -----------------------------------

    ofstream outfile;
    ofstream outfile2;
    stringstream filenamecombine;
    stringstream filenamecombine2;
    
    filenamecombine << "vtkoutput/c_" << step << ".vtk";
    string filename = filenamecombine.str();
    outfile.open(filename.c_str(), std::ios::out);

    // -----------------------------------
    //	Write the 'vtk' file header:
    // -----------------------------------

    string d = "   ";
    outfile << "# vtk DataFile Version 3.1" << endl;
    outfile << "VTK file containing grid data" << endl;
    outfile << "ASCII" << endl;
    outfile << " " << endl;
    outfile << "DATASET STRUCTURED_POINTS" << endl;
    outfile << "DIMENSIONS" << d << nx << d << ny << d << nz << endl;
    outfile << "ORIGIN " << d << 0 << d << 0 << d << 0 << endl;
    outfile << "SPACING" << d << 1.0 << d << 1.0 << d << 1.0 << endl;
    outfile << " " << endl;
    outfile << "POINT_DATA " << nxyz << endl;
    outfile << "SCALARS c float" << endl;
    outfile << "LOOKUP_TABLE default" << endl;

    // -----------------------------------
    //	Write the data:
    // NOTE: x-data increases fastest,
    //       then y-data, then z-data
    // -----------------------------------

    for(size_t k=0;k<nz;k++)
        for(size_t j=0;j<ny;j++)
            for(size_t i=0;i<nx;i++)
            {
                int id = nx*ny*k + nx*j + i;
                double point = c[id];
                //if (point < 1e-10) point = 0.0; // making really small numbers == 0 
                outfile << point << endl;
            }

    // -----------------------------------
    //	Close the file:
    // -----------------------------------

    outfile.close();
    // vtkoutput for water
    // -----------------------------------
    // Define the file location and name:
    // -----------------------------------


    filenamecombine2 << "vtkoutput/w_" << step << ".vtk";
    string filename2 = filenamecombine2.str();
    outfile2.open(filename2.c_str(), std::ios::out);

    // -----------------------------------
    //	Write the 'vtk' file header:
    // -----------------------------------

    outfile2 << "# vtk DataFile Version 3.1" << endl;
    outfile2 << "VTK file containing grid data" << endl;
    outfile2 << "ASCII" << endl;
    outfile2 << " " << endl;
    outfile2 << "DATASET STRUCTURED_POINTS" << endl;
    outfile2 << "DIMENSIONS" << d << nx << d << ny << d << nz << endl;
    outfile2 << "ORIGIN " << d << 0 << d << 0 << d << 0 << endl;
    outfile2 << "SPACING" << d << 1.0 << d << 1.0 << d << 1.0 << endl;
    outfile2 << " " << endl;
    outfile2 << "POINT_DATA " << nxyz << endl;
    outfile2 << "SCALARS w float" << endl;
    outfile2 << "LOOKUP_TABLE default" << endl;

    // -----------------------------------
    //	Write the data:
    // NOTE: x-data increases fastest,
    //       then y-data, then z-data
    // -----------------------------------

    for(size_t k=0;k<nz;k++)
        for(size_t j=0;j<ny;j++)
            for(size_t i=0;i<nx;i++)
            {
                int id = nx*ny*k + nx*j + i;
                double point = water[id];
                // for paraview
                if (point < 1e-30) point = 0.0; // making really small numbers == 0 
                outfile2 << point << endl;
            }

    // -----------------------------------
    //	Close the file:
    // -----------------------------------

    outfile2.close();
    
}



// -------------------------------------------------------------------------
// Run unit tests for this App:
// -------------------------------------------------------------------------

void PFSips::runUnitTests()
{
    bool pass;
    pass = lapKernUnitTest();
    if(pass)
        cout << "\t- lapKernUnitTest -------------- PASSED\n";
    else
        cout << "\t- lapKernUnitTest -------------- FAILED\n";
}



// -------------------------------------------------------------------------
// Unit tests for this App:
// -------------------------------------------------------------------------

bool PFSips::lapKernUnitTest()
{
    // 3X3X3 scalar field with ones except the central node
    double sf[27] = {1,1,1,1,1,1,1,1,1,1,1,1,1,0,1,1,1,1,1,1,1,1,1,1,1,1,1};
    double solution[27] = {0,0,0,0,-1,0,0,0,0,0,-1,0,-1,6,-1,0,-1,0,0,0,0,0,-1,0,0,0,0};
    // allocate space on device
    double* sf_d;
    hipMalloc((void**) &sf_d,27*sizeof(double));
    cudaCheckErrors("hipMalloc fail");
    // copy sf to device
    hipMemcpy(sf_d,sf,27*sizeof(double),hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D fail");
    // launch kernel
    dim3 grid(1,1,3);
    dim3 TpB(32,32,1);
    testLapSIPS<<<grid,TpB>>>(sf_d,3,3,3,1.0,bx,by,bz);
    // copy data back to host
    hipMemcpy(sf,sf_d,27*sizeof(double),hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H fail");
    // print out results
    for(size_t i=0;i<27;i++)
        /* cout << "i=" << i << " sf=" << sf[i] << " sol=" << solution[i] << endl; */
        if( sf[i] != solution[i]) 
        {
            cout << "i=" << i << " sf=" << sf[i] << " sol=" << solution[i] << endl;
            return false;
        }
    return true;
}
