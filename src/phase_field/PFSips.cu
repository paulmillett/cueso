# include <iostream>    // endl
# include <fstream>     // for ofstream
# include <string>      // for string
# include <sstream>     // for stringstream
# include <math.h>
# include "PFSips.h"
# include "PFSipsKernels.h"
# include "../utils/cudaErrorMacros.h" // for cudaCheckErrors & cudaCheckAsyncErrors


using std::string;
using std::stringstream;
using std::cout;
using std::endl;
using std::ofstream;

// -------------------------------------------------------------------------
// Constructor:
// -------------------------------------------------------------------------

PFSips::PFSips(const GetPot& input_params)
    : rng(1234)
{

    // ---------------------------------------
    // Assign variables from 'input_params':
    // ---------------------------------------

    nx = input_params("Domain/nx",1);
    ny = input_params("Domain/ny",1);
    nz = input_params("Domain/nz",1);
    nxyz = nx*ny*nz;
    dx = input_params("Domain/dx",1.0);
    dy = input_params("Domain/dy",1.0);
    dz = input_params("Domain/dz",1.0);
    dt = input_params("Time/dt",1.0);
    bx = input_params("PFSips/bx",0);
    by = input_params("PFSips/by",1);
    bz = input_params("PFSips/bz",1);
    numSteps = input_params("Time/nstep",1);
    co = input_params("PFSips/co",0.20);
    c2 = input_params("PFSips/c2",0.20);
    c3 = input_params("PFSips/c3",0.20);
    r1 = input_params("PFSips/r1",1.0);
    r2 = input_params("PFSips/r2",0.0);
    NS_depth = input_params("PFSips/NS_depth",10);
    M = input_params("PFSips/M",1.0);
    mobReSize = input_params("PFSips/mobReSize",0.35);
    kap = input_params("PFSips/kap",1.0);
    water_CB = input_params("PFSips/water_CB",1.0);
    mobReSize = input_params("PFSips/mobReSize",0.35);
    chiPS = input_params("PFSips/chiPS",0.034);
    chiPN = input_params("PFSips/chiPN",1.5);
    chiCond = input_params("PFSips/chiCond",0);
    chiFreeze = input_params("PFSips/chiFreeze",1.75);
    phiCutoff = input_params("PFSips/phiCutoff",0.75);
    N = input_params("PFSips/N",100.0);
    A = input_params("PFSips/A",1.0);
    Tinit = input_params("PFSips/Tinit",298);
    noiseStr = input_params("PFSips/noiseStr",0.1);
    D0 = input_params("PFSips/D0",1.0);
    nu = input_params("PFSips/nu",1.0);
    gamma = input_params("PFSips/gamma",1.0);
    Mweight = input_params("PFSips/Mweight",100.0);
    Mvolume = input_params("PFSips/Mvolume",0.1);
    numOutputs = input_params("Output/numOutputs",1);
    outInterval = numSteps/numOutputs;
    // ---------------------------------------
    // Set up cuda kernel launch variables:
    // ---------------------------------------

    blockSize.x = input_params("GPU/blockSize.x",0);
    blockSize.y = input_params("GPU/blockSize.y",0);
    blockSize.z = input_params("GPU/blockSize.z",0);

    // set default kernel launch parameters
    if(blockSize.x == 0) blockSize.x = 32;
    if(blockSize.y == 0) blockSize.y = 32;
    if(blockSize.z == 0) blockSize.z = 1;

    // calculate the number of blocks to be used (3-D block grid)
    int totalBlockSize = blockSize.x*blockSize.y*blockSize.z;
    blocks.x = (nx + blockSize.x - 1)/blockSize.x;
    blocks.y = (ny + blockSize.y - 1)/blockSize.y;
    blocks.z = (nz + blockSize.z - 1)/blockSize.z;

    // perform some assumption checking
    int numBlocks = blocks.x*blocks.y*blocks.z;
    int totalNumThreads = numBlocks*totalBlockSize;
    if(totalNumThreads < nxyz)
        throw "GPU Kernel Launch setup lacks sufficient threads!\n";
    if(totalBlockSize > 1024)
        throw "Total number of threads per block exceeds 1024";

}



// -------------------------------------------------------------------------
// Destructor:
// -------------------------------------------------------------------------

PFSips::~PFSips()
{

    // ----------------------------------------
    // free up device memory:
    // ----------------------------------------

    hipFree(c_d);
    hipFree(df_d);
    hipFree(cpyBuff_d);
    hipFree(Mob_d);
    hipFree(nonUniformLap_d);
    hipFree(Mob_d);
    hipFree(devState);
}



// -------------------------------------------------------------------------
// Initialize system:
// -------------------------------------------------------------------------

void PFSips::initSystem()
{
		
    // ----------------------------------------
    // Initialize concentration fields:
    // ----------------------------------------
	 srand(time(NULL));      // setting the seed  
	 // random initialization
    int xHolder = 0;
    int zone1 = r1*(nx-NS_depth); 
    int zone2 = r2*(nx-NS_depth);
    int zone3 = nx - zone1 - zone2 - NS_depth; 
    for(size_t i=0;i<nxyz;i++) {
        double r = (double)rand()/RAND_MAX;
        // create NonSolvent layer
        while (xHolder < NS_depth) 
        {
            c.push_back(0.0);
            xHolder++;
        }
        xHolder = 0;
        // initialize first polymer layer
        while (xHolder < zone1) 
        {
            r = (double)rand()/RAND_MAX; 
            c.push_back(co + 0.1*(r-0.5)); 
            xHolder++;
        }
        xHolder = 0;
        // initialize second polymer layer
        while (xHolder < zone2) 
        {
            r = (double)rand()/RAND_MAX; 
            c.push_back(c2 + 0.1*(r-0.5)); 
            xHolder++;
        }
        xHolder = 0;
        // initialize third polymer layer
        while (xHolder < zone3) 
        {
            r = (double)rand()/RAND_MAX; 
            c.push_back(c3 + 0.1*(r-0.5)); 
            xHolder++;
        }
        xHolder = 0;
    }
    
    // ----------------------------------------
    // Allocate memory on device and copy data
    // and copy data from host to device
    // ----------------------------------------

    // allocate memory on device
    size = nxyz*sizeof(double);
    hipMalloc((void**) &c_d,size);
    cudaCheckErrors("hipMalloc fail");
    hipMalloc((void**) &df_d,size);
    cudaCheckErrors("hipMalloc fail");
    hipMalloc((void**) &cpyBuff_d,size);
    cudaCheckErrors("hipMalloc fail");
    hipMalloc((void**) &Mob_d,size);
    cudaCheckErrors("hipMalloc fail");
    hipMalloc((void**) &nonUniformLap_d,size);
    cudaCheckErrors("hipMalloc fail");
    // allocate memory for cuRAND state
    hipMalloc((void**) &devState,nxyz*sizeof(hiprandState));
    cudaCheckErrors("hipMalloc fail");
    // copy concentration array to device
    hipMemcpy(c_d,&c[0],size,hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D fail");
    
    // ----------------------------------------
    // Initialize thermal fluctuations of
    // polymer concentration
    // ----------------------------------------
    
    init_cuRAND<<<blocks,blockSize>>>(time(NULL),devState,nx,ny,nz);
    
}



// -------------------------------------------------------------------------
// Take one step forward in time:
// -------------------------------------------------------------------------

void PFSips::computeInterval(int interval)
{

    // ----------------------------------------
    //	Set the time step:
    // ----------------------------------------

    current_step = interval*outInterval;

    // ----------------------------------------
    //	Evolve system by solving CH equation:
    // ----------------------------------------

    for(size_t i=0;i<outInterval;i++)
    {
        // calculate the laplacian of c_d and store in df_d
        calculateLapBoundaries<<<blocks,blockSize>>>(c_d,df_d,nx,ny,nz,dx,bx,by,bz); 
        cudaCheckAsyncErrors("calculateLap kernel fail");
        hipDeviceSynchronize();
        
        // calculate the chemical potential and store in df_d
        calculateChemPotFH<<<blocks,blockSize>>>(c_d,df_d,kap,A,water_CB,chiCond,chiPS,chiPN,
        														N,nx,ny,nz,current_step,dt);
        cudaCheckAsyncErrors("calculateChemPotFH kernel fail");
        hipDeviceSynchronize();
        
        // calculate mobility and store it in Mob_d
        calculateMobility<<<blocks,blockSize>>>(c_d,Mob_d,M,mobReSize,nx,ny,nz,phiCutoff,water_CB,
        								        current_step,dt,chiCond,N,gamma,nu,D0,Mweight,Mvolume);
        cudaCheckAsyncErrors("calculateMobility kernel fail");
        hipDeviceSynchronize();
     
        // calculate the laplacian of the chemical potential, then update c_d
        // using an Euler update
        lapChemPotAndUpdateBoundaries<<<blocks,blockSize>>>(c_d,df_d,Mob_d,nonUniformLap_d,
        												    M,dt,nx,ny,nz,dx,bx,by,bz);
        cudaCheckAsyncErrors("lapChemPotAndUpdateBoundaries kernel fail");
        hipDeviceSynchronize();

        // add thermal fluctuations of polymer concentration
        addNoise<<<blocks,blockSize>>>(c_d, nx, ny, nz, dt, current_step, chiCond, water_CB, phiCutoff, devState);
        cudaCheckAsyncErrors("addNoise kernel fail");
        hipDeviceSynchronize();
    }

    // ----------------------------------------
    //	Copy data back to host for writing:
    // ----------------------------------------

    populateCopyBufferSIPS<<<blocks,blockSize>>>(c_d,cpyBuff_d,nx,ny,nz);
    hipMemcpyAsync(&c[0],c_d,size,hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpyAsync D2H fail");
    
}



// -------------------------------------------------------------------------
// Write output:
// -------------------------------------------------------------------------

void PFSips::writeOutput(int step)
{

    // -----------------------------------
    // Define the file location and name:
    // -----------------------------------

    ofstream outfile;
    stringstream filenamecombine;
    filenamecombine << "vtkoutput/c_" << step << ".vtk";
    string filename = filenamecombine.str();
    outfile.open(filename.c_str(), std::ios::out);

    // -----------------------------------
    //	Write the 'vtk' file header:
    // -----------------------------------

    string d = "   ";
    outfile << "# vtk DataFile Version 3.1" << endl;
    outfile << "VTK file containing grid data" << endl;
    outfile << "ASCII" << endl;
    outfile << " " << endl;
    outfile << "DATASET STRUCTURED_POINTS" << endl;
    outfile << "DIMENSIONS" << d << nx << d << ny << d << nz << endl;
    outfile << "ORIGIN " << d << 0 << d << 0 << d << 0 << endl;
    outfile << "SPACING" << d << 1.0 << d << 1.0 << d << 1.0 << endl;
    outfile << " " << endl;
    outfile << "POINT_DATA " << nxyz << endl;
    outfile << "SCALARS c float" << endl;
    outfile << "LOOKUP_TABLE default" << endl;

    // -----------------------------------
    //	Write the data:
    // NOTE: x-data increases fastest,
    //       then y-data, then z-data
    // -----------------------------------

    for(size_t k=0;k<nz;k++)
        for(size_t j=0;j<ny;j++)
            for(size_t i=0;i<nx;i++)
            {
                int id = nx*ny*k + nx*j + i;
                double point = c[id];
                //if (point < 1e-10) point = 0.0; // making really small numbers == 0 
                outfile << point << endl;
            }

    // -----------------------------------
    //	Close the file:
    // -----------------------------------

    outfile.close();
        
}



// -------------------------------------------------------------------------
// Run unit tests for this App:
// -------------------------------------------------------------------------

void PFSips::runUnitTests()
{
    bool pass;
    pass = lapKernUnitTest();
    if(pass)
        cout << "\t- lapKernUnitTest -------------- PASSED\n";
    else
        cout << "\t- lapKernUnitTest -------------- FAILED\n";
}



// -------------------------------------------------------------------------
// Unit tests for this App:
// -------------------------------------------------------------------------

bool PFSips::lapKernUnitTest()
{
    // 3X3X3 scalar field with ones except the central node
    double sf[27] = {1,1,1,1,1,1,1,1,1,1,1,1,1,0,1,1,1,1,1,1,1,1,1,1,1,1,1};
    double solution[27] = {0,0,0,0,-1,0,0,0,0,0,-1,0,-1,6,-1,0,-1,0,0,0,0,0,-1,0,0,0,0};
    // allocate space on device
    double* sf_d;
    hipMalloc((void**) &sf_d,27*sizeof(double));
    cudaCheckErrors("hipMalloc fail");
    // copy sf to device
    hipMemcpy(sf_d,sf,27*sizeof(double),hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D fail");
    // launch kernel
    dim3 grid(1,1,3);
    dim3 TpB(32,32,1);
    testLapSIPS<<<grid,TpB>>>(sf_d,3,3,3,1.0,bx,by,bz);
    // copy data back to host
    hipMemcpy(sf,sf_d,27*sizeof(double),hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H fail");
    // print out results
    for(size_t i=0;i<27;i++)
        /* cout << "i=" << i << " sf=" << sf[i] << " sol=" << solution[i] << endl; */
        if( sf[i] != solution[i]) 
        {
            cout << "i=" << i << " sf=" << sf[i] << " sol=" << solution[i] << endl;
            return false;
        }
    return true;
}
