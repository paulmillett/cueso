#include "hip/hip_runtime.h"
 /*
 * PFSipsKernels.cpp
 * Copyright (C) 2020 M. Rosario Cervellere <rosario.cervellere@gmail.com>
 *
 * Distributed under terms of the MIT license.
 */

#include "PFSipsKernels.h"
#include <stdio.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



// -------------------------------------------------------
// Device Functions
// -------------------------------------------------------


/**********************************************************
   * Laplacian of non-uniform mobility field 
   * for Cahn-Hilliard Euler update
   ********************************************************/

__device__ double laplacianNonUniformMob(double *f, double *Mob,int gid, int x, int y, int z,
                                         int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
	// get id of neighbors for no-flux and PBCs
   int xlid,xrid,ylid,yrid,zlid,zrid;
   // -----------------------------------
   // X-Direction Boundaries
   // -----------------------------------
	if (bX) {
		// PBCs (x-dir.)
		if(x == 0) xlid = nx*ny*z + nx*y + nx-1;
		else xlid = nx*ny*z + nx*y + x-1;
		if(x == nx-1) xrid = nx*ny*z + nx*y + 0;
		else xrid = nx*ny*z + nx*y + x+1;
	}
	else {
	 	// no-flux BC (x-dir.)
		if (x == 0) xlid = nx*ny*z + nx*y + x;
		else xlid = nx*ny*z + nx*y + x-1;
		if (x == nx-1) xrid = nx*ny*z + nx*y + x;
		else xrid = nx*ny*z + nx*y + x+1;
   }
   // -----------------------------------
   // Y-Direction Boundaries
   // -----------------------------------
	if (bY) {
		// PBC Apply
	   if(y == 0) ylid = nx*ny*z + nx*(ny-1) + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*0 + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
   }
   else {
   	// no-flux BC (y-dir.)
      if(y == 0) ylid = nx*ny*z + nx*y + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*y + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
	}
   // -----------------------------------
   // Z-Direction Boundaries
   // -----------------------------------
	if (bZ) {
		// PBC Apply (z-dir.)
   	if(z == 0) zlid = nx*ny*(nz-1) + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*0 + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
   }
	else {
		// no-flux BC (z-dir.)
		if(z == 0) zlid = nx*ny*z + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*z + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
	}

    // ------------------------------------------
    // begin laplacian
	// ------------------------------------------
	
    // get values of neighbors for mobility
    double mobXl = Mob[xlid];
    double mobXr = Mob[xrid];
    double mobYl = Mob[ylid];
    double mobYr = Mob[yrid];
    double mobZl = Mob[zlid];
    double mobZr = Mob[zrid];
    // get values of neighbors for mu
    double xl = f[xlid];
    double xr = f[xrid];
    double yl = f[ylid];
    double yr = f[yrid];
    double zl = f[zlid];
    double zr = f[zrid];
    // get value of current points
    double bo = Mob[gid];
    double fo = f[gid];
    // begin laplacian
    double bx1 = 2.0/(1.0/mobXl + 1.0/bo);
    double bx2 = 2.0/(1.0/mobXr + 1.0/bo);
    double by1 = 2.0/(1.0/mobYl + 1.0/bo);
    double by2 = 2.0/(1.0/mobYr + 1.0/bo);
    double bz1 = 2.0/(1.0/mobZl + 1.0/bo);
    double bz2 = 2.0/(1.0/mobZr + 1.0/bo);
    double lapx = (xl*bx1 + xr*bx2 - (bx1+bx2)*fo)/(h*h); 
    double lapy = (yl*by1 + yr*by2 - (by1+by2)*fo)/(h*h);
    double lapz = (zl*bz1 + zr*bz2 - (bz1+bz2)*fo)/(h*h);
    double lapNonUniform = lapx + lapy + lapz;
    return lapNonUniform;
}   
   

/*********************************************************
   * Compute Laplacian with user specified 
   * boundary conditions (UpdateBoundaries)
   ******************************************************/
	
__device__ double laplacianUpdateBoundaries(double* f,int gid, int x, int y, int z, 
								            int nx, int ny, int nz, double h, 
								            bool bX, bool bY, bool bZ)
{
    // get id of neighbors with periodic boundary conditions
    // and no-flux conditions
    int xlid,xrid,ylid,yrid,zlid,zrid;
    // -----------------------------------
    // X-Direction Boundaries
    // -----------------------------------
    if (bX) {
        // PBCs (x-dir.)
        if(x == 0) xlid = nx*ny*z + nx*y + nx-1;
        else xlid = nx*ny*z + nx*y + x-1;
        if(x == nx-1) xrid = nx*ny*z + nx*y + 0;
        else xrid = nx*ny*z + nx*y + x+1;
    }
    else {
        // no-flux BC (x-dir.)
		if (x == 0) xlid = nx*ny*z + nx*y + x;
		else xlid = nx*ny*z + nx*y + x-1;
		if (x == nx-1) xrid = nx*ny*z + nx*y + x;
		else xrid = nx*ny*z + nx*y + x+1;
    }
    // -----------------------------------
    // Y-Direction Boundaries
    // -----------------------------------
	if (bY) {
        // PBC Apply
        if(y == 0) ylid = nx*ny*z + nx*(ny-1) + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*0 + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
    }
    else {
   	// no-flux BC (y-dir.)
        if(y == 0) ylid = nx*ny*z + nx*y + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*y + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
    }
    // -----------------------------------
    // Z-Direction Boundaries
    // -----------------------------------
	if (bZ) {
		// PBC Apply (z-dir.)
   	if(z == 0) zlid = nx*ny*(nz-1) + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*0 + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
    }
	else {
		// no-flux BC (z-dir.)
		if(z == 0) zlid = nx*ny*z + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*z + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
	}
    // get values of neighbors
    double xl = f[xlid];
    double xr = f[xrid];
    double yl = f[ylid];
    double yr = f[yrid];
    double zl = f[zlid];
    double zr = f[zrid];
    double lap = (xl+xr+yl+yr+zl+zr-6.0*f[gid])/(h*h);
    return lap;
}


/*************************************************************
  * compute chi with linear weighted average
  ***********************************************************/

__device__ double chiDiffuse(double locWater, double chiPS, double chiPN)
{
    double chi = chiPN*locWater + chiPS*(1.0-locWater);
	return chi;
}


/*************************************************************
	* Compute the chemical potential using the 1st derivative
	* of the  binary Flory-Huggins free energy of mixing with
	* respect to c
	*
	* F = c*log(c)/N + (1-c)*log(1-c) + chi*c*(1-c)
	*
	*
	* dF/dc = (log(c) + 1)/N - log(1 - c) - 1.0 
	*         + chi*(1 - 2*c)
	*
	***********************************************************/

__device__ double freeEnergyBiFH(double cc, double chi, double N, double lap_c, double kap, double A)
{
   double c_fh = 0.0;
   if (cc < 0.0) c_fh = 0.0001;
   else if (cc > 1.0) c_fh = 0.999;
   else c_fh = cc;
   double FH = (log(c_fh) + 1.0)/N - log(1.0-c_fh) - 1.0 + chi*(1.0-2.0*c_fh) - kap*lap_c;
   if (cc <= 0.0) FH = -1.5*A*sqrt(-cc) - kap*lap_c;   
   return FH;
}

/*************************************************************
  * Compute second derivative of FH with respect to phi
  ***********************************************************/
  
__device__ double d2dc2_FH(double cc, double N)
{
   double c2_fh = 0.0;
   if (cc < 0.0) c2_fh = 0.0001;
   else if (cc > 1.0) c2_fh = 0.999;
   else c2_fh = cc;
   double FH_2 = 0.5 * (1.0/(N*c2_fh) + 1.0/(1.0-c2_fh));
   return FH_2;	
}

/*************************************************************
  * Compute diffusion coefficient via phillies eq.
  ***********************************************************/

__device__ double philliesDiffusion(double cc, double gamma, double nu, 
								    double D0, double Mweight, double Mvolume)
{
	double cc_d = 1.0;
	double rho = Mweight/Mvolume;
	if (cc >= 1.0) cc_d = 1.0 * rho; // convert phi to g/L	
	else if (cc < 0.0) cc_d = 0.0001 * rho; // convert phi to g/L 
	else cc_d = cc * rho; // convert phi to g/L
	double Dp = D0 * exp(-gamma * pow(cc_d,nu));
	return Dp;
}


// -------------------------------------------------------
// Device Kernels for Testing
// -------------------------------------------------------


/****************************************************************
  * Kernels for unit testing the laplacian devices 
  ***************************************************************/

__global__ void testLapSIPS(double* f, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        f[gid] = laplacianUpdateBoundaries(f,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}

__global__ void testLapNonUniformMob(double* f, double *Mob, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        f[gid] = laplacianNonUniformMob(f,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}




// -------------------------------------------------------
// Device Kernels for Simulation
// -------------------------------------------------------


/*********************************************************
  * Compute the laplacian of the concentration array c and w
  * and store it in the device array df and wdf
  *******************************************************/

__global__ void calculateLapBoundaries(double* c,double* df, int nx, int ny, int nz, 
													double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        df[gid] = laplacianUpdateBoundaries(c,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}




/*********************************************************
  * Computes the chemical potential of a concentration
  * order parameter and stores it in the df_d array.
  *******************************************************/


__global__ void calculateChemPotFH(double* c,double* w,double* df, double kap, double A, double chiPS, double chiPN, double N, int nx, int ny, int nz, int current_step, double dt)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        double cc = c[gid];
        double ww = w[gid];
        double lap_c = df[gid];
        // compute interaction parameter
        double chi = chiDiffuse(ww,chiPS,chiPN);
        // compute chemical potential
        df[gid] = freeEnergyBiFH(cc,chi,N,lap_c,kap,A); 
    }
}


/*********************************************************
  * Computes the mobility of a concentration order
  * parameter and stores it in the Mob_d array.
  *******************************************************/
  
__global__ void calculateMobility(double* c,double* Mob, double M,double mobReSize, int nx, int ny, int nz,
											 double phiCutoff, double N,
        									 double gamma, double nu, double D0, double Mweight, double Mvolume, double Tcast)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        M = 1.0;
        int gid = nx*ny*idz + nx*idy + idx;
        double cc = c[gid];
        double FH2 = d2dc2_FH(cc,N);
        double D_phil = philliesDiffusion(cc,gamma,nu,D0,Mweight,Mvolume);
        double Dtemp = D0*Tcast/273.15;
        M = Dtemp*D_phil/FH2;
        if (M > 1.0) M = 1.0;     // making mobility max = 1
        else if (M < 0.0) M = 0.001; // mobility min = 0.001
        // Using phiCutoff as vitrification
        if (cc > phiCutoff) { 
            M *= 1e-6;
        }
        // resize mobility to be similar to experiments
        M *= mobReSize;
        Mob[gid] = M;		  
    }
}

/************************************************************************************
  * Computes the non-uniform mobility and chemical potential laplacian, multiplies 
  * it by the time step to get the RHS of the CH equation, then uses this RHS value 
  * to perform an Euler update of the concentration in time.
  ***********************************************************************************/

__global__ void lapChemPotAndUpdateBoundaries(double* c,double* df,double* Mob,double* nonUniformLap, double dt, int nx, int ny, int nz, double h,bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        // compute chemical potential laplacain with non-uniform mobility
        // and user defined boundaries (no-flux or PBCs)
        nonUniformLap[gid] = laplacianNonUniformMob(df,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
        c[gid] += nonUniformLap[gid]*dt;
    } 
}



__global__ void calculate_muNS(double*w, double*c, double* muNS, double* Mob, double Dw, double water_CB, double gamma, double nu, double Mweight, double Mvolume, int nx, int ny, int nz)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        
        // calculate mu for NonSolvent NS diffusion
        // make x = 0 coagulation bath composition
        if (idx == 0) w[gid] = water_CB;
        double ww = w[gid];
        // check that polymer < 1.0 and greater than 0.0
        double cc = c[gid];
        if (cc < 0.0) cc = 0.0;
        else if (cc > 1.0) cc = 1.0;
        
        // assign muNS for calculating laplacian
        muNS[gid] =  ww;
        
        double D_NS_phil = philliesDiffusion(cc,gamma,nu,Dw,Mweight,Mvolume);
        Mob[gid] = D_NS_phil;
        if (Mob[gid] < 0.0) Mob[gid] = 0.0;
    }
    
}

__global__ void calculateLapBoundaries_muNS(double* df, double* muNS, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        df[gid] = laplacianUpdateBoundaries(muNS,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}

__global__ void calculateNonUniformLapBoundaries_muNS(double* muNS, double* Mob,double* nonUniformLap, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        nonUniformLap[gid] = laplacianNonUniformMob(muNS,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}

__global__ void update_water(double* w,double* df, double* Mob, double* nonUniformLap, double dt, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // here we're re-using the Mob array for Dw_nonUniform
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        w[gid] += nonUniformLap[gid]*dt;
    }
}


/**********************************************************************
  * initialize cuRAND for thermal fluctuations of polymerconcentration
  *********************************************************************/
__global__ void init_cuRAND(unsigned long seed,hiprandState *state,int nx,int ny,int nz)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        hiprand_init(seed,gid,0,&state[gid]);
    }
}


/************************************************************
  * Add random fluctuations for non-trivial solution (cuRand)
  ***********************************************************/
__global__ void addNoise(double *c,int nx, int ny, int nz, double dt, int current_step, 
                         double water_CB,double phiCutoff,hiprandState *state)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        double noise = hiprand_uniform_double(&state[gid]);
        double cc = c[gid];
        double noiseScale = 1.0;
        // add random fluctuations with euler update
        if (cc > phiCutoff) noise = 0.5; // no fluctuations for phi < 0
        else if (cc < 0.0) noise = 0.5;  // no fluctuations for phi > phiCutoff
        c[gid] += 0.1*(noise-0.5)*dt*noiseScale;
    }
}


/*********************************************************
  * Copies the contents of c into cpyBuffer so the c data
  * can be asynchronously transfered from the device to
  * the host.
  *******************************************************/

__global__ void populateCopyBufferSIPS(double* c,double* cpyBuff, int nx, int ny, int nz)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        // copy the contents of c to cpyBuff
        cpyBuff[gid] = c[gid];
    }
}
