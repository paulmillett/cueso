#include "hip/hip_runtime.h"
/*
 * PFSipsKernels.cpp
 * Copyright (C) 2018 Joseph Carmack <joseph.liping@gmail.com>
 *
 * Distributed under terms of the MIT license.
 */

#include "PFSipsKernels.h"
#include <stdio.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



// -------------------------------------------------------
// Device Functions
// -------------------------------------------------------


/**********************************************************
   * Laplacian of non-uniform mobility field 
   * for Cahn-Hilliard Euler update
   ********************************************************/

__device__ double laplacianNonUniformMob(double *f, double *Mob,int gid, int x, int y, int z,
                                         int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
	// get id of neighbors for no-flux and PBCs
   int xlid,xrid,ylid,yrid,zlid,zrid;
   // -----------------------------------
   // X-Direction Boundaries
   // -----------------------------------
	if (bX) {
		// PBCs (x-dir.)
		if(x == 0) xlid = nx*ny*z + nx*y + nx-1;
		else xlid = nx*ny*z + nx*y + x-1;
		if(x == nx-1) xrid = nx*ny*z + nx*y + 0;
		else xrid = nx*ny*z + nx*y + x+1;
	}
	else {
	 	// no-flux BC (x-dir.)
		if (x == 0) xlid = nx*ny*z + nx*y + x;
		else xlid = nx*ny*z + nx*y + x-1;
		if (x == nx-1) xrid = nx*ny*z + nx*y + x;
		else xrid = nx*ny*z + nx*y + x+1;
   }
   // -----------------------------------
   // Y-Direction Boundaries
   // -----------------------------------
	if (bY) {
		// PBC Apply
	   if(y == 0) ylid = nx*ny*z + nx*(ny-1) + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*0 + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
   }
   else {
   	// no-flux BC (y-dir.)
      if(y == 0) ylid = nx*ny*z + nx*y + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*y + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
	}
   // -----------------------------------
   // Z-Direction Boundaries
   // -----------------------------------
	if (bZ) {
		// PBC Apply (z-dir.)
   	if(z == 0) zlid = nx*ny*(nz-1) + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*0 + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
   }
	else {
		// no-flux BC (z-dir.)
		if(z == 0) zlid = nx*ny*z + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*z + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
	}

    // ------------------------------------------
    // begin laplacian
	// ------------------------------------------
	
    // get values of neighbors for mobility
    double mobXl = Mob[xlid];
    double mobXr = Mob[xrid];
    double mobYl = Mob[ylid];
    double mobYr = Mob[yrid];
    double mobZl = Mob[zlid];
    double mobZr = Mob[zrid];
    // get values of neighbors for mu
    double xl = f[xlid];
    double xr = f[xrid];
    double yl = f[ylid];
    double yr = f[yrid];
    double zl = f[zlid];
    double zr = f[zrid];
    // get value of current points
    double bo = Mob[gid];
    double fo = f[gid];
    // begin laplacian
    double bx1 = 2.0/(1.0/mobXl + 1.0/bo);
    double bx2 = 2.0/(1.0/mobXr + 1.0/bo);
    double by1 = 2.0/(1.0/mobYl + 1.0/bo);
    double by2 = 2.0/(1.0/mobYr + 1.0/bo);
    double bz1 = 2.0/(1.0/mobZl + 1.0/bo);
    double bz2 = 2.0/(1.0/mobZr + 1.0/bo);
    double lapx = (xl*bx1 + xr*bx2 - (bx1+bx2)*fo)/(h*h); 
    double lapy = (yl*by1 + yr*by2 - (by1+by2)*fo)/(h*h);
    double lapz = (zl*bz1 + zr*bz2 - (bz1+bz2)*fo)/(h*h);
    double lapNonUniform = lapx + lapy + lapz;
    return lapNonUniform;
}   
   

/*********************************************************
   * Compute Laplacian with user specified 
   * boundary conditions (UpdateBoundaries)
   ******************************************************/
	
__device__ double laplacianUpdateBoundaries(double* f,int gid, int x, int y, int z, 
								            int nx, int ny, int nz, double h, 
								            bool bX, bool bY, bool bZ)
{
    // get id of neighbors with periodic boundary conditions
    // and no-flux conditions
    int xlid,xrid,ylid,yrid,zlid,zrid;
    // -----------------------------------
    // X-Direction Boundaries
    // -----------------------------------
    if (bX) {
        // PBCs (x-dir.)
        if(x == 0) xlid = nx*ny*z + nx*y + nx-1;
        else xlid = nx*ny*z + nx*y + x-1;
        if(x == nx-1) xrid = nx*ny*z + nx*y + 0;
        else xrid = nx*ny*z + nx*y + x+1;
    }
    else {
        // no-flux BC (x-dir.)
		if (x == 0) xlid = nx*ny*z + nx*y + x;
		else xlid = nx*ny*z + nx*y + x-1;
		if (x == nx-1) xrid = nx*ny*z + nx*y + x;
		else xrid = nx*ny*z + nx*y + x+1;
    }
    // -----------------------------------
    // Y-Direction Boundaries
    // -----------------------------------
	if (bY) {
        // PBC Apply
        if(y == 0) ylid = nx*ny*z + nx*(ny-1) + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*0 + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
    }
    else {
   	// no-flux BC (y-dir.)
        if(y == 0) ylid = nx*ny*z + nx*y + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*y + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
    }
    // -----------------------------------
    // Z-Direction Boundaries
    // -----------------------------------
	if (bZ) {
		// PBC Apply (z-dir.)
   	if(z == 0) zlid = nx*ny*(nz-1) + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*0 + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
    }
	else {
		// no-flux BC (z-dir.)
		if(z == 0) zlid = nx*ny*z + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*z + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
	}
    // get values of neighbors
    double xl = f[xlid];
    double xr = f[xrid];
    double yl = f[ylid];
    double yr = f[yrid];
    double zl = f[zlid];
    double zr = f[zrid];
    double lap = (xl+xr+yl+yr+zl+zr-6.0*f[gid])/(h*h);
    return lap;
}


/*************************************************************
  * Compute diffusive interaction parameter in x-direction
  ***********************************************************/

__device__ double chiDiffuse(double water_CB, double chiPS, double chiPN, double chiCond, int current_step, double dt)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
    double water_diff = (water_CB-0.0)*erfc((idx)/(2.0*sqrt(chiCond*double(current_step)*dt)))+ 0.0;
    double chiPS_diff = chiPN*water_diff + chiPS*(1.0-water_diff);
	return chiPS_diff;
}

__device__ double waterDiff(double water_CB, int current_step, double dt,double chiCond)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
    double water_diff = (water_CB-0.0)*erfc((idx)/(2.0*sqrt(chiCond*double(current_step)*dt)))+ 0.0;
	return water_diff;
}



/*************************************************************
	* Compute the chemical potential using the 1st derivative
	* of the  binary Flory-Huggins free energy of mixing with
	* respect to c
	*
	* F = c*log(c)/N + (1-c)*log(1-c) + chi*c*(1-c)
	*
	*
	* dF/dc = (log(c) + 1)/N - log(1 - c) - 1.0 
	*         + chi*(1 - 2*c)
	*
	***********************************************************/

__device__ double freeEnergyBiFH(double cc, double chiPS_diff, double N, double lap_c, double kap, double A)
{
   double c_fh = 0.0;
   if (cc < 0.0) c_fh = 0.0001;
   else if (cc > 1.0) c_fh = 0.999;
   else c_fh = cc;
   double FH = (log(c_fh) + 1.0)/N - log(1.0-c_fh) - 1.0 + chiPS_diff*(1.0-2.0*c_fh) - kap*lap_c;
   if (cc <= 0.0) FH = -1.5*A*sqrt(-cc) - kap*lap_c;   
   return FH;
}

/*************************************************************
  * Compute second derivative of FH with respect to phi
  ***********************************************************/
  
__device__ double d2dc2_FH(double cc, double N)
{
   double c2_fh = 0.0;
   if (cc < 0.0) c2_fh = 0.0001;
   else if (cc > 1.0) c2_fh = 0.999;
   else c2_fh = cc;
   double FH_2 = 0.5 * (1.0/(N*c2_fh) + 1.0/(1.0-c2_fh));
   return FH_2;	
}

/*************************************************************
  * Compute diffusion coefficient via phillies eq.
  ***********************************************************/

__device__ double philliesDiffusion(double cc, double gamma, double nu, 
								    double D0, double Mweight, double Mvolume)
{
	double cc_d = 1.0;
	double rho = Mweight/Mvolume;
	if (cc >= 1.0) cc_d = 1.0 * rho; // convert phi to g/L	
	else if (cc < 0.0) cc_d = 0.0001 * rho; // convert phi to g/L 
	else cc_d = cc * rho; // convert phi to g/L
	double Dp = D0 * exp(-gamma * pow(cc_d,nu));
	return Dp;
}


// -------------------------------------------------------
// Device Kernels for Testing
// -------------------------------------------------------


/****************************************************************
  * Kernels for unit testing the laplacian devices 
  ***************************************************************/

__global__ void testLapSIPS(double* f, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        f[gid] = laplacianUpdateBoundaries(f,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}

__global__ void testLapNonUniformMob(double* f, double *Mob, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        f[gid] = laplacianNonUniformMob(f,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}




// -------------------------------------------------------
// Device Kernels for Simulation
// -------------------------------------------------------


/*********************************************************
  * Compute the laplacian of the concentration array c
  * and store it in the device array df.
  *******************************************************/

__global__ void calculateLapBoundaries(double* c,double* df, int nx, int ny, int nz, 
													double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        df[gid] = laplacianUpdateBoundaries(c,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}



/*********************************************************
  * Computes the chemical potential of a concentration
  * order parameter and stores it in the df_d array.
  *******************************************************/


__global__ void calculateChemPotFH(double* c,double* df, double kap, double A, double water_CB,
                                   double chiCond, double chiPS, double chiPN, double N, 
                                   int nx, int ny, int nz, int current_step, double dt)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        double cc = c[gid];
        double lap_c = df[gid];
        // compute interaction parameter
        double chi = chiDiffuse(water_CB,chiPS,chiPN,chiCond,current_step,dt);
        // compute chemical potential
        df[gid] = freeEnergyBiFH(cc,chi,N,lap_c,kap,A); 
    }
}


/*********************************************************
  * Computes the mobility of a concentration order
  * parameter and stores it in the Mob_d array.
  *******************************************************/
  
__global__ void calculateMobility(double* c, double* Mob, double M,double mobReSize, int nx, int ny, int nz,
											 double phiCutoff,double water_CB, int current_step, double dt,double chiCond, double N,
        									 double gamma, double nu, double D0, double Mweight, double Mvolume)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        M = 1.0;
        int gid = nx*ny*idz + nx*idy + idx;
        double cc = c[gid];
        double FH2 = d2dc2_FH(cc,N);
        double D_phil = philliesDiffusion(cc,gamma,nu,D0,Mweight,Mvolume);
        M = D0*D_phil/FH2;
        if (M > 1.0) M = 1.0;     // making mobility max = 1
        else if (M < 0.0) M = 0.001; // mobility min = 0.001
        // exponential decrease in mobility 
        // after phiCutoff has been reached
        if (cc > phiCutoff) { 
            double xNorm = (cc - phiCutoff)/(1.0 - phiCutoff);
            double mobScale = 1.0*exp(-10.0*xNorm); // 
            M *= mobScale;
        }
        // ---------------------------------------------------------
        // TODO 
        // scaling mobility based on water concentration
        // use lower diffusion instead...?
        // ---------------------------------------------------------
        // testing mobility scaling with water concentration
        //double water_cutoff = waterDiff(water_CB,current_step,dt,chiCond);
        /*if (water_cutoff > 0.30) {
            double xWnorm = (water_cutoff - 0.30)/(water_CB-0.30);
            double waterScale = 1.0 - (0+((1.0 - 0.0)/(1.0 + exp(-10.0*(xWnorm-(0.0+1.0)/2)))));
            M *= waterScale;
        }*/
        M *= mobReSize;
        Mob[gid] = M;		  
    }
}

/************************************************************************************
  * Computes the non-uniform mobility and chemical potential laplacian, multiplies 
  * it by the time step to get the RHS of the CH equation, then uses this RHS value 
  * to perform an Euler update of the concentration in time.
  ***********************************************************************************/

__global__ void lapChemPotAndUpdateBoundaries(double* c,double* df,double* Mob,double* nonUniformLap,
                                              double M, double dt, int nx, int ny, int nz, double h, 
                                              bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        // compute chemical potential laplacain with non-uniform mobility
        // and user defined boundaries (no-flux or PBCs)
        nonUniformLap[gid] = laplacianNonUniformMob(df,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);    
        c[gid] += nonUniformLap[gid]*dt;
    }
}


/**********************************************************************
  * initialize cuRAND for thermal fluctuations of polymerconcentration
  *********************************************************************/
__global__ void init_cuRAND(unsigned long seed,hiprandState *state,int nx,int ny,int nz)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        hiprand_init(seed,gid,0,&state[gid]);
    }
}


/************************************************************
  * Add random fluctuations for non-trivial solution (cuRand)
  ***********************************************************/
__global__ void addNoise(double *c,int nx, int ny, int nz, double dt, int current_step,double chiCond, 
                         double water_CB,double phiCutoff,hiprandState *state)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        double water_cutoff = waterDiff(water_CB,current_step,dt,chiCond);
        double noise = hiprand_uniform_double(&state[gid]);
        double cc = c[gid];
        double noiseScale = 1.0;
        // add random fluctuations with euler update
        if (cc > phiCutoff) noise = 0.5; // no fluctuations for phi < 0
        else if (cc < 0.0) noise = 0.5;  // no fluctuations for phi > phiCutoff
        // ------------------------------------------------------
        // TODO scaling noise
        // ------------------------------------------------------
        // need to minimise noise effect on vitrified morphology
        // testing different methods
        // ------------------------------------------------------
        // stepwise decrease in noise
        // if (water_cutoff > 0.3) noise = 0.5;  
        // scaling noise based off of water concentration 
        // using a simple exponential function
        /*if (water_cutoff > 0.3) {
            double noise_xNorm = (water_cutoff - 0.3)/(water_CB-0.3);
            double noiseScale = 1.0 * exp(-10.0*noise_xNorm);
        }*/
        // scaling noise similar to mobility scaling
        /*if (water_cutoff > 0.30) {
            double xWnorm = (water_cutoff - 0.30)/(water_CB-0.30);
            double noiseScale = 1.0 - (0+((1.0 - 0.0)/(1.0 + exp(-10.0*(xWnorm-(0.0+1.0)/2)))));
        }*/
        c[gid] += 0.1*(noise-0.5)*dt*noiseScale;
    }
}


/*********************************************************
  * Copies the contents of c into cpyBuffer so the c data
  * can be asynchronously transfered from the device to
  * the host.
  *******************************************************/

__global__ void populateCopyBufferSIPS(double* c,double* cpyBuff, int nx, int ny, int nz)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        // copy the contents of c to cpyBuff
        cpyBuff[gid] = c[gid];
    }
}
